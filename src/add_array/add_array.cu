#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "../cuda_common/cuda_check.h"

__global__ void add_array_kernel(float *A, float *B, float *C, int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
    {
        C[i] = A[i] + B[i];
    }
}

extern "C" void add_array(float *A, float *B, float *C, int N, int grid, int block)
{

    float *d_a, *d_b, *d_c;

    CUDA_CHECK(hipMalloc(&d_a, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_b, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_c, N * sizeof(float)));

    CUDA_CHECK(hipMemcpy(d_a, A, N * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b, B, N * sizeof(float), hipMemcpyHostToDevice));

    add_array_kernel<<<grid, block>>>(d_a, d_b, d_c, N);

    CUDA_CHECK(hipMemcpy(C, d_c, N * sizeof(float), hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(d_a));
    CUDA_CHECK(hipFree(d_b));
    CUDA_CHECK(hipFree(d_c));
}