#include <hip/hip_runtime.h>

#include "../cuda_common/cuda_check.h"

extern "C" void cuda_device_reset()
{
    CUDA_CHECK(hipDeviceReset());
}

extern "C" void cuda_set_device()
{
    CUDA_CHECK(hipSetDevice(0));
}