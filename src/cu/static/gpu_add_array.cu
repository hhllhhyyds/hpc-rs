
#include <hip/hip_runtime.h>
extern "C" __global__ void add_array(float *A, float *B, float *C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
    {
        C[i] = A[i] + B[i];
    } 
}