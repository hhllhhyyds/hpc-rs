#include <hip/hip_runtime.h>

__global__ void
conv_2d_basic_kernel(const float *in, float *out, int width, int height, const float *filter, int r)
{
    const int out_xi = blockIdx.x * blockDim.x + threadIdx.x;
    const int out_yi = blockIdx.y * blockDim.y + threadIdx.y;

    const int filter_edge = 2 * r + 1;

    float value = 0.0;

    for (int i = 0; i < filter_edge; ++i)
    {
        for (int j = 0; j < filter_edge; ++j)
        {
            int in_xi = out_xi - r + i;
            int in_yi = out_yi - r + j;
            if (in_xi >= 0 && in_xi < width && in_yi >= 0 && in_yi < height)
            {
                value += filter[j * filter_edge + i] * in[in_yi * width + in_xi];
            }
        }
    }

    out[out_yi * width + out_xi] = value;
}

/// @param input device pointer
/// @param out device pointer
/// @param filter device pointer
extern "C" void conv_2d_basic(const float *input, float *out, const int width, const int height, const float *filter, const int r)
{
    dim3 block(32, 32);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);
    conv_2d_basic_kernel<<<grid, block>>>(input, out, width, height, filter, r);
}