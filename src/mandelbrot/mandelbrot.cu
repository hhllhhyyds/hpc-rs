#include "hip/hip_runtime.h"
#include "../cuda_common/cuda_check.h"

struct C_MandelbrotGenConfig
{
    double x_range_start;
    double x_range_end;
    double y_range_start;
    double y_range_end;
    int x_pixel_count;
    int y_pixel_count;
    double diverge_limit;
    int iter_count_limit;
};

__constant__ C_MandelbrotGenConfig d_config;

__global__ void gen_mandelbrot_set_kernel(unsigned int *iter_count)
{
    int xi = blockIdx.x * blockDim.x + threadIdx.x;
    int yi = blockIdx.y * blockDim.y + threadIdx.y;

    if (xi < d_config.x_pixel_count && yi < d_config.y_pixel_count)
    {
        const double c_x = d_config.x_range_start + (d_config.x_range_end - d_config.x_range_start) * xi / d_config.x_pixel_count;
        const double c_y = d_config.y_range_start + (d_config.y_range_end - d_config.y_range_start) * yi / d_config.y_pixel_count;

        double z_x = c_x;
        double z_y = c_y;

        const int pos = yi * d_config.x_pixel_count + xi;
        iter_count[pos] = d_config.iter_count_limit;

        const double limit = d_config.diverge_limit * d_config.diverge_limit;
        for (int count = 0; count < d_config.iter_count_limit; ++count)
        {
            double re = z_x * z_x - z_y * z_y + c_x;
            double im = 2.0 * z_x * z_y + c_y;
            z_x = re;
            z_y = im;
            if ((z_x * z_x + z_y * z_y) > limit)
            {
                iter_count[pos] = count + 1;
                break;
            }
        }
    }
}

extern "C" void gen_mandelbrot_set(unsigned int *set, const struct C_MandelbrotGenConfig *config)
{
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_config), config, sizeof(C_MandelbrotGenConfig)));

    dim3 block(32, 32);
    dim3 grid((config->x_pixel_count + block.x - 1) / block.x, (config->y_pixel_count + block.y - 1) / block.y);

    const int data_size = config->x_pixel_count * config->y_pixel_count * sizeof(unsigned int);

    unsigned int *d_set;
    CUDA_CHECK(hipMalloc((void **)&d_set, data_size));

    gen_mandelbrot_set_kernel<<<grid, block>>>(d_set);

    CUDA_CHECK(hipMemcpy(set, d_set, data_size, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipFree(d_set));
}